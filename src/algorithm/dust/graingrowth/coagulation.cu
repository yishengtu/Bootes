#include "hip/hip_runtime.h"
#include "coagulation.hpp"
#include "graingrowthmodel.hpp"
#include "../../util/util.hpp"
#include "../../index_def.hpp"
#include "../../mesh/mesh.hpp"
#include "../../../defs.hpp"
//#include "../terminalvel.hpp"
#include <cmath>

#define CHECK_CUDA_ERROR(val) check( (val), #val, __FILE__, __LINE__)
template <class T>
void check(T err, const char * const errStr, const char * const file, const int line) {
    if(err != hipSuccess) {
	    cerr << "CUDA error at: " << file << ":" << line << endl;
	    cerr << hipGetErrorString(err) << " : " << errStr << endl;
	    exit(1);
    }
}

struct meshsim
{
double *Phi_grav_x1surface;
double *Phi_grav_x2surface;
double *Phi_grav_x3surface;

double *dx1p;
double *dx2p;
double *dx3p;
meshsim(int size)
{
   int state = 0;
   hipMalloc(&Phi_grav_x1surface, size*sizeof(double));
   hipMalloc(&Phi_grav_x2surface, size*sizeof(double));
   hipMalloc(&Phi_grav_x3surface, size*sizeof(double));
   hipMalloc(&dx1p, size*sizeof(double));
   hipMalloc(&dx2p, size*sizeof(double));
   hipMalloc(&dx3p, size*sizeof(double));
   //hipMemcpy(Phi_grav_x1surface, &state, sizeof(double), hipMemcpyHostToDevice);
}

~meshsim()
{
   hipFree(Phi_grav_x1surface);
   hipFree(Phi_grav_x2surface);
   hipFree(Phi_grav_x3surface);
   hipFree(dx1p);
   hipFree(dx2p);
   hipFree(dx3p);
}

/*
__device__ void lock() {
   while(atomicCAS(mutex,0,1) != 0);
}

__device__ void unlock() {
   atomicExch(mutex, 0);
}
*/
};


__device__ void grain_growth_model_stick(double &s1, double &s2, double &dv, double res[2]){
    res[0] = dv * M_PI * pow((s1 + s2), 2.0);
    res[1] = 0.0;
}

__device__ void cu_dust_terminalvelocityapprixmation_xyz(double &vg1, double &vg2, double &vg3,
                                           double &g1,  double &g2,  double &g3,
                                           double &rhod, double &ts,
                                           double &pd1, double &pd2, double &pd3){
    pd1 = rhod * vg1 + g1 * ts;
    pd2 = rhod * vg2 + g2 * ts;
    pd3 = rhod * vg3 + g3 * ts;
    }

__global__ void growth(double *dcons, double *prim, double *stoppingtimemesh, meshsim grav, double dt, double NUMSPECIES, double dminDensity, int *shape,
		int x1s, int x1l, int x2s, int x2l, int x3s, int x3l,
                //double *grain_number_array, double *grain_vr_array, double *grain_vtheta_array,
                //double *grain_vphi_array, double *num_here, double *Mmat,
		double *d_GrainSizeList_arr, double *d_GrainMassList_arr);

__device__ void grain_growth_one_cell(double *num,
                           double *vr, double *vtheta, double *vphi, double *num_here, double *Mmat,
                           double *grain_size_list, double *grain_mass_list, double dt, int NUM_SPECIES){
    double dt_here = dt;
    double temp;
    int index = threadIdx.x;// + blockIdx.x * blockDim.x;
    int stride = blockDim.x;// * gridDim.x;

    for (int i = index; i < NUM_SPECIES; i+=stride) {num_here[i] = num[i];}
    //__syncthreads();

    bool redo = true;
    double dt_tot = 0;
    while (redo || dt_tot < dt)
    {
        redo = false;   // set it to false first
        // BootesArray<double> Mmat; Mmat.NewBootesArray(NUM_SPECIES);         // initialize with 0s

        for (int k = index; k < NUM_SPECIES; k+=stride){
            Mmat[k] = 0;
        }
	  //__syncthreads();

        for (int j = index; j < NUM_SPECIES; j+=stride){
            for (int k = j; k < NUM_SPECIES; ++ k){
		//if (k < j) continue;
                double dv_bulk = sqrt(pow((vr[j] - vr[k]), 2) + pow((vtheta[j] - vtheta[k]), 2) + pow((vphi[j] - vphi[k]), 2));
                // double dv_vortex = dv_ormel(grain_size_list[j], grain_size_list[k], rhogas, tempgas);
                double dv = dv_bulk;

                double KL1[2];                  //K1 = KL1[0], L1 = KL1[1]
                double KL2[2];
                grain_growth_model_stick(grain_size_list[j], grain_size_list[k], dv, KL1);
                grain_growth_model_stick(grain_size_list[k], grain_size_list[j], dv, KL2);

                // replace the for statement here with new method, use V1.8's for if needed
                // gain via coagulation
                double numjtimesnumk = num_here[j] * num_here[k];
                int cog_res = 0;        // the following loop is equivalent to searchsorted in numpy
                while (cog_res < NUM_SPECIES){
                    if (grain_mass_list[cog_res] >= (grain_mass_list[j] + grain_mass_list[k])){
                        break;
                    }
                    cog_res += 1;
                }
                if (j == k){
                    if (cog_res == NUM_SPECIES){
                        Mmat[cog_res - 1] += 0.5 * (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1] * numjtimesnumk;
                    }
                    else{
                        double eps = (grain_mass_list[j] + grain_mass_list[k] - grain_mass_list[cog_res - 1]) / (grain_mass_list[cog_res] - grain_mass_list[cog_res - 1]);
                        Mmat[cog_res]     += 0.5 * (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res] * numjtimesnumk * eps;
                        Mmat[cog_res - 1] += 0.5 * (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1] * numjtimesnumk * (1.0 - eps);
                    }
                }
                else{
                    if (cog_res == NUM_SPECIES){
                        temp = (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1]* numjtimesnumk;
			//Mmat[cog_res - 1] += temp;
			atomicAdd ( Mmat[cog_res - 1],temp);
                    //    Mmat[cog_res - 1] += (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1]* numjtimesnumk;
                    }
                    else{
                        double eps =  (grain_mass_list[j] + grain_mass_list[k] - grain_mass_list[cog_res - 1]) / (grain_mass_list[cog_res] - grain_mass_list[cog_res - 1]);
                        Mmat[cog_res]     += (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res]* numjtimesnumk * eps;
                        Mmat[cog_res - 1] += (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1]* numjtimesnumk * (1.0 - eps);
                    }
                }
                // gain via fragmentation
                Mmat[0] += KL1[1] * grain_mass_list[k] / grain_mass_list[0] * numjtimesnumk;
                Mmat[0] += KL2[1] * grain_mass_list[j] / grain_mass_list[0] * numjtimesnumk;
                // lost via coagulation and fragmentation
                Mmat[k] -= (KL1[0] + KL1[1]) * numjtimesnumk;
                if (k != j){
                    Mmat[j] -= (KL2[0] + KL2[1]) * numjtimesnumk;
                }
                // break if there is a problem
                /*
                if (isnan(Mmat(0))){
                }
                if (isnan(Mmat(j))){
                }
                if (isnan(Mmat(k))){
                }
                */
            }
        }
        //__syncthreads();

        dt_tot += dt_here;
        for (int i = index; i < NUM_SPECIES; i+=stride){
            num_here[i] += Mmat[i] * dt_here;
            if (num_here[i] < 0)
            {
                num_here[i] = 0;
            }
        }
	  //__syncthreads();

        //cout << dt_tot << '\t' << min_dt << '\t' << dt << '\n' << flush;
    }
      //__syncthreads();

    for (int i = index; i < NUM_SPECIES; i+=stride){num[i] = num_here[i];}
}


void grain_growth(mesh &m, BootesArray<double> &stoppingtimemesh, double &dt){
    int NUMSPECIES = m.NUMSPECIES;
    double dminDensity = m.dminDensity;
    //double *grain_number_array = new double[NUMSPECIES];
    //double *grain_vr_array     = new double[NUMSPECIES];
    //double *grain_vtheta_array = new double[NUMSPECIES];
    //double *grain_vphi_array   = new double[NUMSPECIES];
    double *grain_number_array;
    double *grain_vr_array;
    double *grain_vtheta_array;
    double *grain_vphi_array;
    double *num_here;
    double *Mmat;
    //hipError_t cerr;

    size_t nspbytes = NUMSPECIES*sizeof(double);
    int device_id;
    
    CHECK_CUDA_ERROR(hipGetDevice(&device_id));

    CHECK_CUDA_ERROR(hipMallocManaged(&grain_number_array, nspbytes));
    //std::cout<<"cerr "<<cerr<<std::endl;
    CHECK_CUDA_ERROR(hipMallocManaged(&grain_vr_array, nspbytes));
    CHECK_CUDA_ERROR(hipMallocManaged(&grain_vtheta_array, nspbytes));
    CHECK_CUDA_ERROR(hipMallocManaged(&grain_vphi_array, nspbytes));
    CHECK_CUDA_ERROR(hipMallocManaged(&num_here, nspbytes));
    CHECK_CUDA_ERROR(hipMallocManaged(&Mmat, nspbytes));
    
    int NG=2; 
    int size1=m.dcons.shape()[4]-2*NG, size2=m.dcons.shape()[3]-2*NG, size3=m.dcons.shape()[2]-2*NG;
    int ncell = m.dcons.shape()[4]*m.dcons.shape()[3]*m.dcons.shape()[2];

    //std::cout<<"size1 "<<size1<<" size2 "<<size2<<" size3 "<<size3<<std::endl;
    int BLKX=32, BLKY=32, BLKZ=32;
    int MB=32;
    size_t grnbytes = m.GrainSizeList.shape()[0]*sizeof(double);
    double *d_GrainSizeList_arr;
    double *d_GrainMassList_arr;
    double *d_stoppingtimemesh;
    size_t stpbytes = stoppingtimemesh.shape()[0]*stoppingtimemesh.shape()[1]*stoppingtimemesh.shape()[2]*stoppingtimemesh.shape()[3]*sizeof(double);
    CHECK_CUDA_ERROR(hipMalloc(&d_GrainSizeList_arr, grnbytes));
    CHECK_CUDA_ERROR(hipMalloc(&d_GrainMassList_arr, grnbytes));
    CHECK_CUDA_ERROR(hipMalloc(&d_stoppingtimemesh, stpbytes));
    CHECK_CUDA_ERROR(hipMemcpy(d_GrainSizeList_arr, m.GrainSizeList.data(), grnbytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_GrainMassList_arr, m.GrainMassList.data(), grnbytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_stoppingtimemesh, stoppingtimemesh.data(), stpbytes, hipMemcpyHostToDevice));

    meshsim grav=meshsim(ncell);
    CHECK_CUDA_ERROR(hipMemcpy(grav.Phi_grav_x1surface,m.grav->Phi_grav_x1surface.data(),ncell*sizeof(double),hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(grav.Phi_grav_x2surface,m.grav->Phi_grav_x2surface.data(),ncell*sizeof(double),hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(grav.Phi_grav_x3surface,m.grav->Phi_grav_x3surface.data(),ncell*sizeof(double),hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(grav.dx1p,m.dx1p.data(),ncell*sizeof(double),hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(grav.dx2p,m.dx2p.data(),ncell*sizeof(double),hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(grav.dx3p,m.dx3p.data(),ncell*sizeof(double),hipMemcpyHostToDevice));
    

    double *d_dcons = (double*)acc_deviceptr(m.dcons.data());
    double *d_prim = (double*)acc_deviceptr(m.prim.data());

    int *shape_m = m.dcons.shape();
    dim3 BlocksperGrid(size1/BLKX+1,size2/BLKY+1, size3/BLKZ+1);
    dim3 ThreadsperBlock(BLKX,BLKY,BLKZ); 
    int x1s = m.x1s;int x1l = m.x1l;int x2s = m.x2s;int x2l = m.x2l;int x3s = m.x3s;int x3l = m.x3l;
    int sharedbytes = 6*NUMSPECIES*sizeof(double); 
    //std::cout<<"x1s "<<x1s<<" x1l "<<x1l<<" x2s "<<x2s<<" x2l "<<x2l<<" x3s "<<x3s<<" x3l "<<x3l<<std::endl;
    growth<<<BlocksperGrid, ThreadsperBlock, sharedbytes>>>(d_dcons, d_prim, d_stoppingtimemesh, grav, dt, NUMSPECIES, dminDensity, shape_m, x1s, x1l, x2s, x2l, x3s, x3l, d_GrainSizeList_arr, d_GrainMassList_arr);
    hipDeviceSynchronize();

}

__global__ void growth(double *dcons, double *prim, double *stoppingtimemesh, meshsim grav, double dt, double NUMSPECIES,double dminDensity, int *shape,
		                int x1s, int x1l, int x2s, int x2l, int x3s, int x3l,
		//double *grain_number_array, double *grain_vr_array, double *grain_vtheta_array, 
		//double *grain_vphi_array, double *num_here, double *Mmat,
		double *GrainSizeList_arr, double *GrainMassList_arr){

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    //int index = blockIdx.x;
    int stridex = blockDim.x * gridDim.x;
    int stridey = blockDim.y * gridDim.y;
    int stridez = blockDim.z * gridDim.z;
    int index_x = threadIdx.x + blockDim.x*blockIdx.x;
    int index_y = threadIdx.y + blockDim.y*blockIdx.y;
    int index_z = threadIdx.z + blockDim.z*blockIdx.z;
    //size_t nspbytes = NUMSPECIES*sizeof(double);
    
    index_x = blockId.x;
    index_y = blockId.y;
    index_z = blockId.z;

    stridex = gridDim.x;
    stridey = gridDim.y;
    stridez = gridDim.z;

    int device_id;
    int size1 = shape[4];
    int size2 = shape[3];
    int size3 = shape[2];
    int size4 = shape[1];
    
    __shared__ grain_number_array[];
    __shared__ grain_vr_array[];
    __shared__ grain_vtheta_array[];
    __shared__ grain_vphi_array[];
    __shared__ num_here[];
    __shared__ Mmat[];

    //hipGetDevice(&device_id);

    for (int kk = index_z+x3s; kk < x3l; kk +=stridez){
        for (int jj = index_y+x2s; jj <x2l; jj +=stridey){
            for (int ii = index_x+x1s; ii < x1l; ii +=stridex){
    
    /*int kk = index_z+x3s;
    int jj = index_y+x2s;
    int ii = index_x+x1s;
    if (kk >= x3l) return;
    if (jj >= x2l) return;
    if (ii >= x1l) return;
    */
                /*
                if (rhomesh(j)[i] < 4e-17){         // if density is below some threshold, just skip coagulation calculations.
                    continue;
                }
                */
                for (int specIND = theadId.x; specIND < NUMSPECIES; specIND +=blockDim.x){
                    //double gas_rho = m.dcons(specIND, IDN, kk, jj, ii);
	            //std::cout<<"grain_number_array "<<grain_number_array[specIND]<<std::endl;
		    int idx_IDN = ii + size1*(jj + size2*(kk + size3 * (IDN + size4 * specIND)));
		    int idx_IM1 = ii + size1*(jj + size2*(kk + size3 * (IM1 + size4 * specIND)));
		    int idx_IM2 = ii + size1*(jj + size2*(kk + size3 * (IM2 + size4 * specIND)));
		    int idx_IM3 = ii + size1*(jj + size2*(kk + size3 * (IM3 + size4 * specIND)));
                    grain_number_array[specIND] = dcons[idx_IDN] / GrainMassList_arr[specIND];
                    grain_vr_array[specIND]     = dcons[idx_IM1] / dcons[idx_IDN];
                    grain_vtheta_array[specIND] = dcons[idx_IM2] / dcons[idx_IDN];
                    grain_vphi_array[specIND]   = dcons[idx_IM3] / dcons[idx_IDN];
		    
                    //cout << m.dcons(specIND, IM1, kk, jj, ii) << '\t';
                }
                __syncthreads();
		//    hipMemPrefetchAsync(grain_number_array, nspbytes, device_id);
		//    hipMemPrefetchAsync(grain_vr_array, nspbytes, device_id);
		//    hipMemPrefetchAsync(grain_vtheta_array, nspbytes, device_id);
		//    hipMemPrefetchAsync(grain_vphi_array, nspbytes, device_id);
		//    hipMemPrefetchAsync(num_here, nspbytes, device_id);
		//    hipMemPrefetchAsync(Mmat, nspbytes, device_id);
                //cout << endl << flush;

              
                grain_growth_one_cell(grain_number_array,
                                      grain_vr_array, grain_vtheta_array, grain_vphi_array, num_here, Mmat,
                                      GrainSizeList_arr, GrainMassList_arr, dt, NUMSPECIES);
		__syncthreads();
                // copy 1-cell results from grain_number_array to m.dcons

                for (int specIND = 0; specIND < NUMSPECIES; specIND ++) {
		    int idx_IDN = ii + size1*(jj + size2*(kk + size3 * (IDN + size4 * specIND)));
		    int idx_IM1 = ii + size1*(jj + size2*(kk + size3 * (IM1 + size4 * specIND)));
		    int idx_IM2 = ii + size1*(jj + size2*(kk + size3 * (IM2 + size4 * specIND)));
		    int idx_IM3 = ii + size1*(jj + size2*(kk + size3 * (IM3 + size4 * specIND)));

                    int idx_IV1 = ii + size1*(jj + size2*(kk + size3 * IV1)); 
                    int idx_IV2 = ii + size1*(jj + size2*(kk + size3 * IV2)); 
                    int idx_IV3 = ii + size1*(jj + size2*(kk + size3 * IV3)); 
		    
		    int idx_stop= ii + size1*(jj + size2*(kk + size3 * specIND));

		    if (grain_number_array[specIND] * GrainMassList_arr[specIND] < dminDensity) {
                        dcons[idx_IDN] = dminDensity;
                        double rhogradphix1;
                        double rhogradphix2;
                        double rhogradphix3;
                        #ifdef ENABLE_GRAVITY
			int kji = ii + size1*(jj + size2*kk);
			int kji1 = ii + 1 + size1*(jj + size2*kk);
			int kj1i = ii + size1*(jj + 1 + size2*kk);
			int k1ji = ii + size1*(jj + (size2*kk+1));

                        rhogradphix1 = dcons[idx_IDN] * (grav.Phi_grav_x1surface[kji1] - grav.Phi_grav_x1surface[kji]) / grav.dx1p[kji];
                        rhogradphix2 = dcons[idx_IDN] * (grav.Phi_grav_x2surface[kj1i] - grav.Phi_grav_x2surface[kji]) / grav.dx2p[kji];
                        rhogradphix3 = dcons[idx_IDN] * (grav.Phi_grav_x3surface[k1ji] - grav.Phi_grav_x3surface[kji]) / grav.dx3p[kji];
                        //rhogradphix1 = dcons[idx_IDN] * (m.grav->Phi_grav_x1surface(kk, jj, ii + 1) - m.grav->Phi_grav_x1surface(kk, jj, ii)) / m.dx1p(kk, jj, ii);
                        //rhogradphix2 = dcons[idx_IDN] * (m.grav->Phi_grav_x2surface(kk, jj + 1, ii) - m.grav->Phi_grav_x2surface(kk, jj, ii)) / m.dx2p(kk, jj, ii);
                        //rhogradphix3 = dcons[idx_IDN] * (m.grav->Phi_grav_x3surface(kk + 1, jj, ii) - m.grav->Phi_grav_x3surface(kk, jj, ii)) / m.dx3p(kk, jj, ii);
                        #else   // set gravity to zero
                        rhogradphix1 = 0;
                        rhogradphix2 = 0;
                        rhogradphix3 = 0;
                        #endif // ENABLE_GRAVITY
                        #ifdef CARTESIAN_COORD
                        cu_dust_terminalvelocityapprixmation_xyz(prim[idx_IV1], prim[idx_IV2], prim[idx_IV3],
                                                              rhogradphix1, rhogradphix2, rhogradphix3,
                                                              dcons[idx_IDN], stoppingtimemesh[idx_stop],
                                                              dcons[idx_IM1], dcons[idx_IM2], dcons[idx_IM3]
                                                              );
                        #endif // CARTESIAN_COORD
                        #ifdef SPHERICAL_POLAR_COORD
                        dust_terminalvelocityapprixmation_rtp(m.prim(IV1, kk, jj, ii), m.prim(IV2, kk, jj, ii), m.prim(IV3, kk, jj, ii),
                                                              rhogradphix1, rhogradphix2, rhogradphix3,
                                                              m.dcons(specIND, IDN, kk, jj, ii), stoppingtimemesh(specIND, kk, jj, ii), m.x1v(ii), m.geo_cot(jj),
                                                              m.dcons(specIND, IM1, kk, jj, ii), m.dcons(specIND, IM2, kk, jj, ii), m.dcons(specIND, IM3, kk, jj, ii)
                                                              );
                        #endif // SPHERICAL_POLAR_COORD
                        //#ifdef DEBUG
                        //std::cout << "drho < 0:\t" << specIND << '\t' << kk << '\t' << jj << '\t' << ii << '\t'
                        //          << m.dcons(specIND, IDN, kk, jj, ii) << '\t' << m.dcons(specIND, IM1, kk, jj, ii) << '\t'
                        //          << m.dcons(specIND, IM2, kk, jj, ii) << '\t' << m.dcons(specIND, IM3, kk, jj, ii) << std::endl << flush;
                        //#endif // DEBUG
                    }
                    else {
                        dcons[idx_IDN] = grain_number_array[specIND] * GrainMassList_arr[specIND];
                        dcons[idx_IM1] = dcons[idx_IDN] * grain_vr_array[specIND];
                        dcons[idx_IM2] = dcons[idx_IDN] * grain_vtheta_array[specIND];
                        dcons[idx_IM3] = dcons[idx_IDN] * grain_vphi_array[specIND];
                    }
                }	
    //        }
    //    }
   // }
    /*
    delete[] grain_number_array;
    delete[] grain_vr_array;
    delete[] grain_vtheta_array;
    delete[] grain_vphi_array;
    */
}
