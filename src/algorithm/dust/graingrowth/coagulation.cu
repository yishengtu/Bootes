#include "hip/hip_runtime.h"
#include "coagulation.hpp"
#include "graingrowthmodel.hpp"
#include "../../util/util.hpp"
#include "../../index_def.hpp"
#include "../../mesh/mesh.hpp"
#include "../../../defs.hpp"
#include "../terminalvel.hpp"
#include <cmath>

__device__ void grain_growth_model_stick(double &s1, double &s2, double &dv, double res[2]){
    res[0] = dv * M_PI * pow((s1 + s2), 2.0);
    res[1] = 0.0;
}


__global__ void grain_growth_one_cell(double *num,
                           double *vr, double *vtheta, double *vphi, double *num_here, double *Mmat,
                           double *grain_size_list, double *grain_mass_list, double dt, int NUM_SPECIES){
    double dt_here = dt;

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < NUM_SPECIES; i+=stride) {num_here[i] = num[i];}
    __syncthreads();

    bool redo = true;
    double dt_tot = 0;
    while (redo || dt_tot < dt)
    {
        redo = false;   // set it to false first
        // BootesArray<double> Mmat; Mmat.NewBootesArray(NUM_SPECIES);         // initialize with 0s

        for (int k = index; k < NUM_SPECIES; k+=stride){
            Mmat[k] = 0;
        }
	  __syncthreads();

        for (int j = index; j < NUM_SPECIES; j+=stride){
            for (int k = j; k < NUM_SPECIES; ++ k){
		//if (k < j) continue;
                double dv_bulk = sqrt(pow((vr[j] - vr[k]), 2) + pow((vtheta[j] - vtheta[k]), 2) + pow((vphi[j] - vphi[k]), 2));
                // double dv_vortex = dv_ormel(grain_size_list[j], grain_size_list[k], rhogas, tempgas);
                double dv = dv_bulk;

                double KL1[2];                  //K1 = KL1[0], L1 = KL1[1]
                double KL2[2];
                grain_growth_model_stick(grain_size_list[j], grain_size_list[k], dv, KL1);
                grain_growth_model_stick(grain_size_list[k], grain_size_list[j], dv, KL2);

                // replace the for statement here with new method, use V1.8's for if needed
                // gain via coagulation
                double numjtimesnumk = num_here[j] * num_here[k];
                int cog_res = 0;        // the following loop is equivalent to searchsorted in numpy
                while (cog_res < NUM_SPECIES){
                    if (grain_mass_list[cog_res] >= (grain_mass_list[j] + grain_mass_list[k])){
                        break;
                    }
                    cog_res += 1;
                }
                if (j == k){
                    if (cog_res == NUM_SPECIES){
                        Mmat[cog_res - 1] += 0.5 * (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1] * numjtimesnumk;
                    }
                    else{
                        double eps = (grain_mass_list[j] + grain_mass_list[k] - grain_mass_list[cog_res - 1]) / (grain_mass_list[cog_res] - grain_mass_list[cog_res - 1]);
                        Mmat[cog_res]     += 0.5 * (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res] * numjtimesnumk * eps;
                        Mmat[cog_res - 1] += 0.5 * (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1] * numjtimesnumk * (1.0 - eps);
                    }
                }
                else{
                    if (cog_res == NUM_SPECIES){
                        Mmat[cog_res - 1] += (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1]* numjtimesnumk;
                    }
                    else{
                        double eps =  (grain_mass_list[j] + grain_mass_list[k] - grain_mass_list[cog_res - 1]) / (grain_mass_list[cog_res] - grain_mass_list[cog_res - 1]);
                        Mmat[cog_res]     += (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res]* numjtimesnumk * eps;
                        Mmat[cog_res - 1] += (KL1[0] * grain_mass_list[k] + KL2[0] * grain_mass_list[j]) / grain_mass_list[cog_res - 1]* numjtimesnumk * (1.0 - eps);
                    }
                }
                // gain via fragmentation
                Mmat[0] += KL1[1] * grain_mass_list[k] / grain_mass_list[0] * numjtimesnumk;
                Mmat[0] += KL2[1] * grain_mass_list[j] / grain_mass_list[0] * numjtimesnumk;
                // lost via coagulation and fragmentation
                Mmat[k] -= (KL1[0] + KL1[1]) * numjtimesnumk;
                if (k != j){
                    Mmat[j] -= (KL2[0] + KL2[1]) * numjtimesnumk;
                }
                // break if there is a problem
                /*
                if (isnan(Mmat(0))){
                }
                if (isnan(Mmat(j))){
                }
                if (isnan(Mmat(k))){
                }
                */
            }
        }
        __syncthreads();

        dt_tot += dt_here;
        for (int i = index; i < NUM_SPECIES; i+=stride){
            num_here[i] += Mmat[i] * dt_here;
            if (num_here[i] < 0)
            {
                num_here[i] = 0;
            }
        }
	  __syncthreads();

        //cout << dt_tot << '\t' << min_dt << '\t' << dt << '\n' << flush;
    }
      __syncthreads();

    for (int i = index; i < NUM_SPECIES; i+=stride){num[i] = num_here[i];}
}


void grain_growth(mesh &m, BootesArray<double> &stoppingtimemesh, double &dt){
    int NUMSPECIES = m.NUMSPECIES;
    //double *grain_number_array = new double[NUMSPECIES];
    //double *grain_vr_array     = new double[NUMSPECIES];
    //double *grain_vtheta_array = new double[NUMSPECIES];
    //double *grain_vphi_array   = new double[NUMSPECIES];
    double *grain_number_array;
    double *grain_vr_array;
    double *grain_vtheta_array;
    double *grain_vphi_array;
    double *num_here;
    double *Mmat;
    

    size_t nspbytes = NUMSPECIES*sizeof(double);
    
    int device_id;
    
    hipGetDevice(&device_id);

    hipMallocManaged(&grain_number_array, nspbytes);
    hipMallocManaged(&grain_vr_array, nspbytes);
    hipMallocManaged(&grain_vtheta_array, nspbytes);
    hipMallocManaged(&grain_vphi_array, nspbytes);
    hipMallocManaged(&num_here, nspbytes);
    hipMallocManaged(&Mmat, nspbytes);

	for (int kk = m.x3s; kk < m.x3l; kk ++){
        for (int jj = m.x2s; jj < m.x2l; jj ++){
            for (int ii = m.x1s; ii < m.x1l; ii ++){
                /*
                if (rhomesh(j)[i] < 4e-17){         // if density is below some threshold, just skip coagulation calculations.
                    continue;
                }
                */
                for (int specIND = 0; specIND < m.NUMSPECIES; specIND ++){
                    //double gas_rho = m.dcons(specIND, IDN, kk, jj, ii);
                    grain_number_array[specIND] = m.dcons(specIND, IDN, kk, jj, ii) / m.GrainMassList(specIND);
                    grain_vr_array[specIND]     = m.dcons(specIND, IM1, kk, jj, ii) / m.dcons(specIND, IDN, kk, jj, ii);
                    grain_vtheta_array[specIND] = m.dcons(specIND, IM2, kk, jj, ii) / m.dcons(specIND, IDN, kk, jj, ii);
                    grain_vphi_array[specIND]   = m.dcons(specIND, IM3, kk, jj, ii) / m.dcons(specIND, IDN, kk, jj, ii);
                    //cout << m.dcons(specIND, IM1, kk, jj, ii) << '\t';
                }
		    hipMemPrefetchAsync(grain_number_array, nspbytes, device_id);
		    hipMemPrefetchAsync(grain_vr_array, nspbytes, device_id);
		    hipMemPrefetchAsync(grain_vtheta_array, nspbytes, device_id);
		    hipMemPrefetchAsync(grain_vphi_array, nspbytes, device_id);
		    hipMemPrefetchAsync(num_here, nspbytes, device_id);
		    hipMemPrefetchAsync(Mmat, nspbytes, device_id);
                //cout << endl << flush;
                size_t grnbytes = m.GrainSizeList.shape()[0]*sizeof(double);
		double *d_GrainSizeList_arr;
		double *d_GrainMassList_arr;
		hipMalloc(&d_GrainSizeList_arr, grnbytes);
		hipMalloc(&d_GrainMassList_arr, grnbytes);
                hipMemcpy(d_GrainSizeList_arr, m.GrainSizeList.data(), grnbytes, hipMemcpyHostToDevice);
                hipMemcpy(d_GrainMassList_arr, m.GrainMassList.data(), grnbytes, hipMemcpyHostToDevice);
                int MB=1;
                grain_growth_one_cell<<<NUMSPECIES/MB, MB>>>(grain_number_array,
                                      grain_vr_array, grain_vtheta_array, grain_vphi_array, num_here, Mmat,
                                      d_GrainSizeList_arr, d_GrainMassList_arr, dt, m.NUMSPECIES);
                // copy 1-cell results from grain_number_array to m.dcons

                for (int specIND = 0; specIND < m.NUMSPECIES; specIND ++) {
                    if (grain_number_array[specIND] * m.GrainMassList(specIND) < m.dminDensity) {
                        m.dcons(specIND, IDN, kk, jj, ii) = m.dminDensity;
                        double rhogradphix1;
                        double rhogradphix2;
                        double rhogradphix3;
                        #ifdef ENABLE_GRAVITY
                        rhogradphix1 = m.dcons(specIND, IDN, kk, jj, ii) * (m.grav->Phi_grav_x1surface(kk, jj, ii + 1) - m.grav->Phi_grav_x1surface(kk, jj, ii)) / m.dx1p(kk, jj, ii);
                        rhogradphix2 = m.dcons(specIND, IDN, kk, jj, ii) * (m.grav->Phi_grav_x2surface(kk, jj + 1, ii) - m.grav->Phi_grav_x2surface(kk, jj, ii)) / m.dx2p(kk, jj, ii);
                        rhogradphix3 = m.dcons(specIND, IDN, kk, jj, ii) * (m.grav->Phi_grav_x3surface(kk + 1, jj, ii) - m.grav->Phi_grav_x3surface(kk, jj, ii)) / m.dx3p(kk, jj, ii);
                        #else   // set gravity to zero
                        rhogradphix1 = 0;
                        rhogradphix2 = 0;
                        rhogradphix3 = 0;
                        #endif // ENABLE_GRAVITY
                        #ifdef CARTESIAN_COORD
                        dust_terminalvelocityapprixmation_xyz(m.prim(IV1, kk, jj, ii), m.prim(IV2, kk, jj, ii), m.prim(IV3, kk, jj, ii),
                                                              rhogradphix1, rhogradphix2, rhogradphix3,
                                                              m.dcons(specIND, IDN, kk, jj, ii), stoppingtimemesh(specIND, kk, jj, ii),
                                                              m.dcons(specIND, IM1, kk, jj, ii), m.dcons(specIND, IM2, kk, jj, ii), m.dcons(specIND, IM3, kk, jj, ii)
                                                              );
                        #endif // CARTESIAN_COORD
                        #ifdef SPHERICAL_POLAR_COORD
                        dust_terminalvelocityapprixmation_rtp(m.prim(IV1, kk, jj, ii), m.prim(IV2, kk, jj, ii), m.prim(IV3, kk, jj, ii),
                                                              rhogradphix1, rhogradphix2, rhogradphix3,
                                                              m.dcons(specIND, IDN, kk, jj, ii), stoppingtimemesh(specIND, kk, jj, ii), m.x1v(ii), m.geo_cot(jj),
                                                              m.dcons(specIND, IM1, kk, jj, ii), m.dcons(specIND, IM2, kk, jj, ii), m.dcons(specIND, IM3, kk, jj, ii)
                                                              );
                        #endif // SPHERICAL_POLAR_COORD
                        #ifdef DEBUG
                        std::cout << "drho < 0:\t" << specIND << '\t' << kk << '\t' << jj << '\t' << ii << '\t'
                                  << m.dcons(specIND, IDN, kk, jj, ii) << '\t' << m.dcons(specIND, IM1, kk, jj, ii) << '\t'
                                  << m.dcons(specIND, IM2, kk, jj, ii) << '\t' << m.dcons(specIND, IM3, kk, jj, ii) << std::endl << flush;
                        #endif // DEBUG
                    }
                    else {
                        m.dcons(specIND, IDN, kk, jj, ii) = grain_number_array[specIND] * m.GrainMassList(specIND);
                        m.dcons(specIND, IM1, kk, jj, ii) = m.dcons(specIND, IDN, kk, jj, ii) * grain_vr_array[specIND];
                        m.dcons(specIND, IM2, kk, jj, ii) = m.dcons(specIND, IDN, kk, jj, ii) * grain_vtheta_array[specIND];
                        m.dcons(specIND, IM3, kk, jj, ii) = m.dcons(specIND, IDN, kk, jj, ii) * grain_vphi_array[specIND];
                    }
                }
            }
        }
    }
    /*
    delete[] grain_number_array;
    delete[] grain_vr_array;
    delete[] grain_vtheta_array;
    delete[] grain_vphi_array;
    */
}
